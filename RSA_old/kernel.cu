#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <string.h>
#include <ctype.h>
#include <stdlib.h>
#include <time.h>
#include <conio.h>
#define MAX 6 // 48 * 32 integers - 1536 bit max
#define THREAD_NUMBER 5
#define INT_SIZE 32

unsigned int	    
		
		ZERO_MEMORY[MAX],
		
		KKK[33] = { 3, 5, 17, 23, 27, 29, 39, 47, 57, 65, 71, 75, 77, 83, 93, 99, 105, 107, 113, 117, 129, 135, 143, 149, 153, 159, 167, 173, 185, 189, 195, 199, 203 },
		HHH[33] = { 285, 297, 299, 303, 309, 315, 323, 327, 329, 339, 353, 359, 363, 365, 369, 383, 387, 395, 413, 419, 429, 437, 453, 465, 467, 479, 483, 485, 489, 497, 507, 509, 513 },

		Base[THREAD_NUMBER],
		Base_[THREAD_NUMBER],

		N_Me,					
		Me,						
		InverseByModuleM_Me,	
		InverseByModuleMMMe,	
		input[THREAD_NUMBER],	
		input_[THREAD_NUMBER],
		N[THREAD_NUMBER] ,
		N_[THREAD_NUMBER] ,
		Mi[THREAD_NUMBER][MAX],     // Large
		Mi_[THREAD_NUMBER][MAX],    // Large
		M[ MAX ],					//
		M_[ MAX ],					//
		$N[MAX],
		MiInv[THREAD_NUMBER] ,
		Mi_Inv[THREAD_NUMBER] ,
		SQR_M_MOD_N_RNS[THREAD_NUMBER],
		SQR_M_MOD_N_RNS_[THREAD_NUMBER] ,
		InverseByModuleMM_[THREAD_NUMBER] ,
		InverseByModuleNM[THREAD_NUMBER] ,
		Mi_InMe[THREAD_NUMBER] ,
		MiInMe [THREAD_NUMBER],
		M_InBase[THREAD_NUMBER],
		MiInBase_[THREAD_NUMBER][THREAD_NUMBER],
		Mi_InBase[THREAD_NUMBER][THREAD_NUMBER],
		ABmodN[2][THREAD_NUMBER];

	bool power[ MAX * INT_SIZE ];
	bool e[ MAX * INT_SIZE ];
	bool d[ MAX * INT_SIZE ];
	
	
	int iterationsCounter,
		lastValueOfStateInformation,
		currentOperationIndex,
		eBitsCount, 
		numberOfIterationsForE,
		dBitsCount, 
		numberOfIterationsForD;

int profiler_Inversions = 0, 
	profiler_extended_euclid_Iterations[1000000], 
	profiler_eeI_counter = -1;


void clearLongVariable ( unsigned int *a ) {
	
	memcpy( a, ZERO_MEMORY, MAX * sizeof(int) );

}

void copyVariable ( unsigned int *source, unsigned int *dest ) {
	
	memcpy( dest, source, MAX * sizeof(int) );

}

// зсунути 32-бітне число вліво на 1 біт і повернути carry flag
int shl (unsigned int &input, int rightBit){
	
	int CF;
	CF = ( input & ( 1 << INT_SIZE - 1 ) ) == ( 1 << INT_SIZE - 1 );
	input <<= 1;             
	input &= 0xFFFFFFFE;	
	input |= rightBit;		
	return CF;
}

// зсунути 32-бітне число вправо на 1 біт і повернути carry flag
int shr (unsigned int &input, int leftBit){
	
	int CF;
	CF =  input & 0x1 ;  
	input >>= 1;
		
	if ( leftBit == 1 ) { 
		input |= 0x80000000;	
	} else { 
		input &= 0x7FFFFFFF;	
	}
	
	return CF;
}

// зсунути 64-бітне число вправо на 1 біт і повернути carry flag
int shr_long (unsigned long long &input, int leftBit){
	
	int CF;
	CF =  input & 0x1 ;  
	input >>= 1;
		
	if ( leftBit == 1 ) { 
		input |= 0x8000000000000000;	
	} else { 
		input &= 0x7FFFFFFFFFFFFFFF;	
	}
	
	return CF;
}

//зсунути "довге число" вліво на 1 біт і повернути carry flag
int shiftToLeftVariable ( unsigned int *input ) {
	int CF = 0;
	for (int i = MAX - 1; i >= 0; i--) {
		CF = shl( input[ i ], CF );
	}
	return CF;
}

//зсунути "довге число" вправо на 1 біт і повернути carry flag
int shiftToRightVariable ( unsigned int *input ) {
	int CF = 0;
	for (int i = 0; i < MAX; i++) {
		CF = shr( input[ i ], CF );
	}

	return CF;
}

void convertToNormalForm (char input[], unsigned int *a){
	
	clearLongVariable (a);

	int temp = 0;
    for( int i = MAX - 1, 
		 int j = strlen(input) - 1, 
		 int p = 0; 
					j >= 0; j-- ) {
		
			if ( input[j] == 32 ) { continue; } // дозволено пробіл для зручності
			else if ( toupper(input[j]) == 65 ) { temp |= 0xA << p; }
			else if ( toupper(input[j]) == 66 ) { temp |= 0xB << p; }
			else if ( toupper(input[j]) == 67 ) { temp |= 0xC << p; }
			else if ( toupper(input[j]) == 68 ) { temp |= 0xD << p; }
			else if ( toupper(input[j]) == 69 ) { temp |= 0xE << p; }
			else if ( toupper(input[j]) == 70 ) { temp |= 0xF << p; }
			else { temp |= ( toupper(input[j]) - 48 ) << p; }
			 
	    p+=4;
		if( p >= 32 || j == 0){ p = 0; a[i] = temp; temp = 0; i--; }
	}

}

void convertFromNormalForm (char output[], unsigned int *a) {
	
	int temp;
	int j = ( MAX * 8 ) - 1;
	output [ j + 1 ] = '\0';
    for( int i = MAX - 1; i >= 0; i-- ) {
		
		int mask = 0xF;

		for ( int p = 0; p < 8; p++) {
			temp = a[ i ] ;
			temp >>= 4*p;
			temp &=  mask ;
			output [ j ] = ( temp > 9 ) ? ( temp + 55 ) : ( temp + 48 );
			j--;
		}
	}
}

void convertPowerToArrayOfBits( unsigned int* $power, bool *power, int *bin_digits_in_power, int *totalIterationsNeeded ){
	
	int c = 0;

	for(int i = MAX * INT_SIZE - 1, bit; i > 0 ; i--){

		bit = shiftToRightVariable($power);
		if (bit == 1){
			*bin_digits_in_power = MAX * INT_SIZE - i;
			c++;
		}
		power[i] = (bit == 1) ? true : false;
	}
	c *= 2;
	c += 2 * (*bin_digits_in_power);


	*totalIterationsNeeded = c;

}




void add ( unsigned int *a,  
		   unsigned int *b,  
		   unsigned int *c) 
{
	int CF = 0;
	
	for (int i = MAX - 1; i >= 0; i--) {

		c[ i ] = a[ i ] + b[ i ] + CF;
		
		if ( ( c[ i ] < a [ i ] ) || ( c[ i ] < b [ i ] ) ) { 
			
			CF = 1; 

		} else {

			CF = 0;
		}
	}
}


void sub ( unsigned int *a,  
		   unsigned int *b,  
		   unsigned int *c) 
{

		
	unsigned int tempResult [MAX]; 
	clearLongVariable ( tempResult );
	
	int CF = 0;

	for (int i = MAX - 1; i >= 0; i--) {
		tempResult[i] = a[i] - b[i] - CF;

		if(b[i] == 0xFFFFFFFF){ 
			CF = 1; 
			continue;
		}
		if(a[i] < (b[i] + CF) ){
			CF = 1;
		} else {
			CF=0;
		}

	}
	copyVariable ( tempResult, c );

}



//
//  1 a > b
//  0 a = b
// -1 a < b
// знакове порівняння чисел
int cmp (unsigned int *a, unsigned int *b) 
{
	
	bool a_positive = !(a[0] & 0x80000000);
	bool b_positive = !(b[0] & 0x80000000);

	// 1. + +
	// 1. + -
	// 1. - +
	// 1. - -

	if ( a_positive && b_positive ){

		for (int i = 0; i < MAX; i++) {
			if ( a[ i ] > b[ i ] ) { 
				return 1; 
			}

			if ( a[ i ] < b[ i ] ) { 
				return -1; 
			}

		}
		return 0;

	} else if(a_positive && !b_positive) {

		return 1;

	} else if(!a_positive && b_positive) {

		return -1;

	} else { // two numbers are negative:
	
		for (int i = 0; i < MAX; i++) {

			if ( a[ i ] > b[ i ] ) { 
				return 1; 
			}
			
			if ( a[ i ] < b[ i ] ) { 
				return -1; 
			}

		}
		return 0;
	}
}




void mod (  unsigned int *a,  
			unsigned int *b, 
			unsigned int *c) 
{
	
		unsigned int temp_A[ MAX ];
		unsigned int R[ MAX ];	// remainder

		copyVariable ( a, temp_A );


		if ( cmp ( temp_A, b) <= 0) {
			copyVariable ( temp_A, c );
			return;
		}

		
		clearLongVariable ( R );   // initialize remainder to zero

		// Integer division (unsigned) with remainder
		for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
			shiftToLeftVariable ( R );					   // left-shift R by 1 bit  
			R[ MAX - 1] |= shiftToLeftVariable ( temp_A ); // set the least-significant bit of R equal to bit i of the numerator

			if ( cmp( R, b ) >= 0 ) {
				sub ( R, b, R );	
			}
		}
		copyVariable ( R, c );   
}




void mod_ ( unsigned int *a,  
			unsigned int b,  
			unsigned int* c) 
{

		unsigned int temp_A[ MAX ];
		unsigned long long R;	// остача

		copyVariable ( a, temp_A );
		
		R = 0;   

		for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
			R <<= 1;					    
			R |= shiftToLeftVariable ( temp_A ); 

			if ( R >= b ) {
				R -= b;	
			}
				
		}

		*c = (unsigned int) R; 

}


// division
void div (  unsigned int *a,  
			unsigned int *b,  
			unsigned int *c) 
{

		unsigned int temp_A[ MAX ];
		unsigned int temp_B[ MAX ];
		unsigned int Q[ MAX ];	// остача 
		unsigned int R[ MAX ];	// залишок 
		unsigned int zero [MAX];
		unsigned int mask = 0x80000000;
		bool aIsLessThenZero = false, 
			 bIsLessThenZero = false;

		for(int i=0; i< MAX;i++){
			zero[i] = 0;
		}

		if (cmp (a,zero) < 0) {
			sub (zero, a, temp_A);
			aIsLessThenZero = true;
		} else {
			copyVariable ( a, temp_A );
		}

		if (cmp (b,zero) < 0) {
			sub (zero, b, temp_B);
			bIsLessThenZero = true;
		} else {
			copyVariable ( b, temp_B );
		}


		clearLongVariable ( Q );   
		clearLongVariable ( R );   
		
		//Integer division (unsigned) with remainder
		//http://en.wikipedia.org/wiki/Division_algorithm
		int j;
		for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
			j =  i / INT_SIZE ;
			shiftToLeftVariable ( R );					    
			R[ MAX - 1] |= shiftToLeftVariable ( temp_A ); 

			if ( cmp( R, temp_B ) >= 0 ) {
			
				sub ( R, temp_B, R );	
				Q[MAX - 1 - j] |= mask;
			}

			mask >>= 1;
			if(mask == 0) mask = 0x80000000;
				
		}

		// - / - = +
		// + / - = -
		// - / + = -
		// + / + = +
		if (aIsLessThenZero ^ bIsLessThenZero) {
			sub (zero, Q, Q);
		}

		copyVariable ( Q, c );   

}


void div_ ( unsigned int *a,  
			unsigned int b,  
			unsigned int *c) 
{

		unsigned int temp_A[ MAX ];
		unsigned long long temp_B;
		unsigned int Q[ MAX ];	// остача 
		unsigned long long R;	// залишок 
		unsigned int zero [MAX];
		bool aIsLessThenZero = false, 
			 bIsLessThenZero = false;

		for(int i=0; i< MAX;i++){
			zero[i] = 0;
		}

		if (cmp (a,zero) < 0) {
			sub (zero, a, temp_A);
			aIsLessThenZero = true;
		} else {
			copyVariable ( a, temp_A );
		}

		if ( b < 0 ) {
			temp_B = b * (-1);
			bIsLessThenZero = true;
		} else {
			temp_B = b;
		}


		clearLongVariable ( Q );  
		R = 0 ;   
		
		unsigned int mask = 0x80000000;
		
		//Integer division (unsigned) with remainder
		//http://en.wikipedia.org/wiki/Division_algorithm
		int j;
		for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
			j =  i / INT_SIZE ;
			R <<= 1;					    
			R |= shiftToLeftVariable ( temp_A ); 

			if (  R >= temp_B ) {
				R -= temp_B;	
				Q[MAX - 1 - j] |= mask;
			}

			mask >>= 1;
			if(mask == 0) mask = 0x80000000;
				
		}

		// - / - = +
		// + / - = -
		// - / + = -
		// + / + = +
		if (aIsLessThenZero ^ bIsLessThenZero) {
			sub (zero, Q, Q);
		}

		copyVariable ( Q, c );   

}




void mul (unsigned int *firstFactor,  
		  unsigned int *secondFactor,  
		  unsigned int *product) 
{

	unsigned int tempFirstFactor[ MAX ]; 
	unsigned int tempSecondFactor[ MAX ];
	unsigned int tempProduct[ MAX ];
	unsigned int zero [MAX];

	bool firstFactorIsLessThenZero = false, 
		 secondFactorIsLessThenZero = false;

	for(int i=0; i < MAX;i++){
		zero[i] = 0;
	}

	if (cmp (firstFactor,zero) < 0) {
		sub (zero, firstFactor, tempFirstFactor);
		firstFactorIsLessThenZero = true;
	} else {
		copyVariable ( firstFactor, tempFirstFactor );
	}

	if (cmp (secondFactor,zero) < 0) {
		sub (zero, secondFactor, tempSecondFactor);
		secondFactorIsLessThenZero = true;
	} else {
		copyVariable ( secondFactor, tempSecondFactor );
	}

	int CF;
	clearLongVariable (product);

	for (int i = 0; i < MAX * INT_SIZE; i++ ) {
		
		CF = shiftToRightVariable ( tempSecondFactor );	

		if ( CF == 1 ) {
			add ( tempFirstFactor, product, product);
		}

		shiftToLeftVariable ( tempFirstFactor );	

	}


	// - / - = +
	// + / - = -
	// - / + = -
	// + / + = +

	copyVariable(product, tempProduct);

	if (firstFactorIsLessThenZero ^ secondFactorIsLessThenZero) {
		sub (zero, product, product);
	}


}



// a * x = -1 mod b
// a * x + b * y = gcd(a,b)
void extended_euclid( unsigned int *a,  
					  unsigned int *b,  
					  unsigned int *x,  
					  unsigned int *y,  
					  unsigned int *d )
{
	unsigned int x1[ MAX ];
	unsigned int x2[ MAX ];
	unsigned int y1[ MAX ];
	unsigned int y2[ MAX ];
	unsigned int q[ MAX ];
	unsigned int r[ MAX ];
	unsigned int zero[ MAX ];
	unsigned int temp[ MAX ];
	unsigned int temp_a[ MAX ];
	unsigned int temp_b[ MAX ];
	
	copyVariable( a, temp_a );
	copyVariable( b, temp_b );
	clearLongVariable ( x1 );
	clearLongVariable ( x2 );
	clearLongVariable ( y1 );
	clearLongVariable ( y2 );
	clearLongVariable ( q );
	clearLongVariable ( r );
	clearLongVariable ( zero );
	clearLongVariable ( temp );

	x2[ MAX-1 ] = 1; // x2 = 1
	x1[ MAX-1 ] = 0; // x1 = 0
	y2[ MAX-1 ] = 0; // y2 = 0
	y1[ MAX-1 ] = 1; // y1 = 1

	profiler_eeI_counter++;
	profiler_extended_euclid_Iterations[profiler_eeI_counter] = 0;

	while ( cmp( temp_b, zero ) > 0) {

		profiler_extended_euclid_Iterations[profiler_eeI_counter]++;

		div ( temp_a, temp_b, q);		//q = a / b, 
		mul ( q, temp_b, temp );
		sub ( temp_a, temp, r);			//r = a - q * b;
		clearLongVariable ( temp );
		mul ( q, x1, temp );
		sub ( x2, temp, x );			//*x = x2 - q * x1, 
		clearLongVariable ( temp );
		mul( q, y1, temp );
		sub( y2, temp, y );				//*y = y2 - q * y1;
		copyVariable( temp_b, temp_a );	//a = b, 
		copyVariable( r, temp_b );		//b = r;
		copyVariable( x1, x2 );			//x2 = x1,  
		copyVariable( x, x1 );			//x1 = *x,  // t
		copyVariable( y1, y2 );			//y2 = y1,  //
		copyVariable( y, y1 );			//y1 = *y;
	 
	}

	copyVariable( temp_a, d );	//*d = a,
	copyVariable( x2, x );		//*x = x2,
	copyVariable( y2, y );		//*y = y2;

}


//input * result = 1 mod module
void InverseByModule( unsigned int *input,  
					  unsigned int *module,  
					  unsigned int *result )
{
	unsigned int x[MAX];
	unsigned int y[MAX];
	unsigned int d[MAX];
	unsigned int zero[MAX];
	
	profiler_Inversions++;

	clearLongVariable(x);
	clearLongVariable(y);
	clearLongVariable(d);
	clearLongVariable(zero);

	extended_euclid(input, module, x, y, d);

	if (cmp ( x, zero ) < 0 ) { 
		add ( x, module, zero );
		copyVariable( zero, result );
	} else {
		copyVariable( x, result );
	}
	//return x < 0 ? x + module : x;
}


void InverseByModule_( unsigned int *input,  
					   unsigned int module,  
					   unsigned int *result )
{
	unsigned int x[MAX];
	unsigned int y[MAX];
	unsigned int d[MAX];
	unsigned int temp[MAX];
	unsigned int zero[MAX];
	
	profiler_Inversions++;

	clearLongVariable(x);
	clearLongVariable(y);
	clearLongVariable(d);
	clearLongVariable(temp);
	clearLongVariable(zero);

	temp[ MAX - 1 ] = module;

	extended_euclid(input, temp, x, y, d);

	if (cmp ( x, zero ) < 0 ) { 
		add ( x, temp, zero );
		 *result = zero[MAX-1];
	} else {
		 *result = x[MAX-1];
	}
}



void getNumberInRNSByModMe( unsigned int* input, unsigned long long* result){

	unsigned int X[MAX];
	unsigned int R[MAX];
	unsigned int tempProduct[MAX];
	unsigned int tempM[MAX];
	unsigned int tempFirstFactor[MAX];
	unsigned long long temp;
	

	for (int i = 0; i < MAX; i++){
		tempM[ i ] = M[ i ];
		X[ i ] = 0;
		R[ i ] = 0;
	}

	// X[i] = INPUT[i] * MiInv[i] * Mi[i]
	for (int i = 0; i < THREAD_NUMBER; i++){
		
		for (int j = 0; j < MAX; j++){
			tempProduct[j] = 0;
			tempFirstFactor [j] = Mi[i][j];
		}

		temp = input[i];
		temp *= MiInv[i];

		while( temp !=0 ){

			if ( shr_long (temp, 0) == 1 ) {
				add ( tempFirstFactor, tempProduct, tempProduct);
			}

			shiftToLeftVariable ( tempFirstFactor );	
		}

		add ( X, tempProduct, X);

	}

	//X[i] % M
	for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
		shiftToLeftVariable ( R );					    
		R[ MAX - 1] |= shiftToLeftVariable ( X );

		if ( cmp( R, tempM ) >= 0 ) {
			sub ( R, tempM, R );	
		}
	}

	//X[i] % M % Me (Me = 2^6)
	*result = 0;
	*result += ( shiftToRightVariable ( R )) == 1 ? 1 : 0;	
	*result += ( shiftToRightVariable ( R )) == 1 ? 2 : 0;	
	*result += ( shiftToRightVariable ( R )) == 1 ? 4 : 0;	
	*result += ( shiftToRightVariable ( R )) == 1 ? 8 : 0;	
	*result += ( shiftToRightVariable ( R )) == 1 ? 16 : 0;
	*result += ( shiftToRightVariable ( R )) == 1 ? 32 : 0;

}

void convertFromRNS( unsigned int* input, unsigned int* result){

	unsigned int X[MAX];
	unsigned int R[MAX];
	unsigned int tempInput[THREAD_NUMBER];
	unsigned int tempProduct[MAX];
	unsigned int tempM[MAX];
	unsigned int tempFirstFactor[MAX];
	unsigned long long temp;
	

	for (int i = 0; i < MAX; i++){
		tempM[ i ] = M[ i ];
		X[ i ] = 0;
		R[ i ] = 0;
	}

	
	// X[i] = INPUT[i] * MiInv[i] * Mi[i]
	for (int i = 0; i < THREAD_NUMBER; i++){
		
		for (int j = 0; j < MAX; j++){
			tempProduct[j] = 0;
			tempFirstFactor [j] = Mi[i][j];
		}

		tempInput[i] = input[i];

		temp = tempInput[i];
		temp *= MiInv[i];

		while( temp !=0 ){

			if ( shr_long (temp, 0) == 1 ) {
				add ( tempFirstFactor, tempProduct, tempProduct);
			}

			shiftToLeftVariable ( tempFirstFactor );	
		}

		add ( X, tempProduct, X);

	}

	//X[i] % M
	for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
		shiftToLeftVariable ( R );					   
		R[ MAX - 1] |= shiftToLeftVariable ( X ); 

		if ( cmp( R, tempM ) >= 0 ) {
			sub ( R, tempM, R );	
		}
	}
	

	mod(R, $N, result);


}

void convertFromRNS_( unsigned int* input, unsigned int* result){

	unsigned int X[MAX];
	unsigned int R[MAX];
	unsigned int tempProduct[MAX];
	unsigned int tempInput[THREAD_NUMBER];
	unsigned int tempM[MAX];
	unsigned int tempFirstFactor[MAX];
	unsigned long long temp;
	

	for (int i = 0; i < MAX; i++){
		tempM[ i ] = M_[ i ];
		X[ i ] = 0;
		R[ i ] = 0;
	}

	

	// X[i] = INPUT[i] * MiInv[i] * Mi[i]
	for (int i = 0; i < THREAD_NUMBER; i++){
		
		for (int j = 0; j < MAX; j++){
			tempProduct[j] = 0;
			tempFirstFactor [j] = Mi_[i][j];
		}

		tempInput[i] = input[i];
		
		temp = tempInput[i];
		temp *= Mi_Inv[i];

		while( temp !=0 ){

			if ( shr_long (temp, 0) == 1 ) {
				add ( tempFirstFactor, tempProduct, tempProduct);
			}

			shiftToLeftVariable ( tempFirstFactor );	
		}

		add ( X, tempProduct, X);

	}

	//X[i] % M
	for (int i = MAX * INT_SIZE - 1; i >= 0 ; i-- ) {
		
		shiftToLeftVariable ( R );					  
		R[ MAX - 1] |= shiftToLeftVariable ( X ); 

		if ( cmp( R, tempM ) >= 0 ) {
			sub ( R, tempM, R );	
		}
	}
	

	mod(R, $N, result);


}

void printCurrentStateInformation(){

	
	//system("cls");
	if( currentOperationIndex == 0 ){
		if( lastValueOfStateInformation != iterationsCounter * 100 /  numberOfIterationsForE ){
			lastValueOfStateInformation = iterationsCounter * 100 /  numberOfIterationsForE;
			printf("Encryption... %i %% done.\n", iterationsCounter * 100 /  numberOfIterationsForE );	
		}

	} else if( currentOperationIndex == 1 ){
		if( lastValueOfStateInformation != iterationsCounter * 100 / numberOfIterationsForD ){
			lastValueOfStateInformation = iterationsCounter * 100 / numberOfIterationsForD;
			printf("Decryption... %i %% done.\n", iterationsCounter * 100 /  numberOfIterationsForD );	
		}

	}

	iterationsCounter++ ;

}




void MM( unsigned int* A, 
		 unsigned int* A_, 
		 unsigned int* B, 
		 unsigned int* B_,
		 unsigned int* R, 
		 unsigned int* R_ )
{

	unsigned long long Q[THREAD_NUMBER];
	unsigned long long Xi[THREAD_NUMBER];
	unsigned long long Q_Me, R_Me, A_Me, B_Me, A_Me2, B_Me2 ;
	unsigned long long Q_[THREAD_NUMBER]; 
	unsigned long long s, s1, s2, currSum, r; 
	unsigned long long Sig[THREAD_NUMBER];
	unsigned long long Beta, temp;


	unsigned long long tempA[THREAD_NUMBER];
	unsigned long long tempB[THREAD_NUMBER];

	int rightBit;

	clock_t begin;
	begin = clock();


	//computation of Q
	for (int i=0; i<THREAD_NUMBER; i++){

		Q[i] = ( Base[ i ]  - A[ i ] ) % Base[i];
		Q[i] *= (B[ i ] % Base[i]);
		Q[i] %= Base[i];
		Q[i] *= (InverseByModuleNM[i] % Base[i]);
		Q[i] %= Base[i] ;
	}

	

	// First Base extension:
	//****************************************************************************

	for (int i=0; i<THREAD_NUMBER; i++){
		Sig [i] = Q[i] % Base [i];
		Sig [i] *= MiInv[i] % Base [i];
		Sig [i] %= Base [i];
	}

	

	for (int i=0; i<THREAD_NUMBER; i++){
		s=0;
		for (int j=0; j<THREAD_NUMBER; j++){
			s += ((MiInBase_[i][j] % Base_ [i]) * (Sig[j] % Base_ [i])) % Base_ [i];
		}


		Q_[i] = s % Base_ [i];
	}

	

	//Extra modulus computation;
	// Only first thread:
	s=0;

	for (int i=0; i<THREAD_NUMBER; i++){
		s += ((MiInMe[i] % Me) * ((Q[i] * MiInv[i]) %  Base [i]) % Me ) % Me;

	}

	Q_Me = s % Me;


	for (int i=0; i<THREAD_NUMBER; i++){
		tempA[i] = A[i];
		tempB[i] = B[i];
	}





	unsigned long long jjj = 1;
	unsigned long long j;
	unsigned int ttempA, ttempB;

	for (int i=0; i<THREAD_NUMBER; i++){

		InverseByModule_(Mi[i], 64, &ttempA);

		

		

		jjj *= ttempA;
	}
		
	
	A_Me2 = jjj % Me;

	

	getNumberInRNSByModMe(A, &A_Me);
	getNumberInRNSByModMe(B, &B_Me);

	//printf("\n[%4.2f] Q_Me, A_Me, B_Me\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	// Computing R in Base_:

	for (int i=0; i<THREAD_NUMBER; i++){ 
		temp = ( (Q_[i] * N_[i]) % Base_[i] + ((unsigned long long)A_[i] * (unsigned long long)B_[i]) % Base_[i] ) % Base_[i] ;
		temp *= InverseByModuleMM_[i] % Base_[i];
		temp %=  Base_[i];
		R_[i] = (unsigned int)temp;

	}

	R_Me = (((A_Me * B_Me)  + (Q_Me * N_Me) )  * InverseByModuleMMMe ) % Me ;

	//printf("\n[%4.2f] R_, R_Me\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	//Second base extension:
	//*********************************************************************************************

	for (int i=0; i<THREAD_NUMBER; i++){

		Xi [i] = R_[i]  % Base_ [i];
		Xi [i] *= Mi_Inv[i]  % Base_ [i];
		Xi [i] %= Base_ [i];
	}


	//Computing beta:
	s=0;
	for (int j=0; j<THREAD_NUMBER; j++){
		s +=   Xi [j] * Mi_InMe[j];
	}

	Beta = ( ((s - R_Me) % Me) * InverseByModuleM_Me) % Me;

	//printf("\n[%4.2f] Beta\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	for (int i=0; i<THREAD_NUMBER; i++){
		s1=0; 
		s2=0; 
		currSum=0; //[s1, s2] 128 bit max;

		for (int j=0; j<THREAD_NUMBER; j++){

			s2 += Mi_InBase[i][j] * Xi[j];

			//overflow
			if(s2 < currSum ) {
				s1++;
			} 

			currSum = s2;

		}


		if ( s1==0 ){

			R[i] = (  s2 - (( Beta % Base[i] ) * M_InBase[i] ) % Base[i]    ) % Base[i];

		} else {

			r = 0;

			s2 -= (( Beta % Base[i] ) * M_InBase[i] ) % Base[i] ;

			for (int j = 4 * INT_SIZE - 1; j > 0 ; j-- ) {

				rightBit = ( s2 & ( 0x8000000000000000 ) ) == ( 0x8000000000000000 ) ? 1 : 0; 
				r <<= 1;
				s2 <<=1;
				s1 <<= 1;            
				s1 &= 0xFFFFFFFFFFFFFFFE;	
				s1 |= rightBit;		
				r |= ( s1 & ( 0x8000000000000000 ) ) == ( 0x8000000000000000 ) ? 1 : 0; 

				if ( r > Base[i] ) {
					r -= Base[i];	
				}
			}

			R[i] = r;

		}

	}

	//printf("\n[%4.2f] R\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	//printCurrentStateInformation();

	//(A * B * M^-1 mod N + Beta * N) mod M
}



void showProfilerInfo(){
printf("\nInversions:			%i", profiler_Inversions);
}



void generateStaticData(){

	clock_t begin;
	unsigned int temp[ MAX ];
	unsigned int $SQR_M_MOD_N[ MAX ];
	unsigned int $InverseByModuleMM_[ MAX ];
	unsigned int $InverseByModule$NM[ MAX ];

	begin = clock();
	// M = Base[0] * Base[1] * ... * Base[n]
	// M_ = Base_[0] * Base_[1] * ... * Base_[n]
	convertToNormalForm( "1" , M); 
	convertToNormalForm( "1" , M_); 

	for(int i=0; i < THREAD_NUMBER; i++){
		clearLongVariable( temp );
		temp[ MAX - 1 ] = Base[i];
		mul ( M, temp, M );

		temp[ MAX - 1 ] = Base_[i];
		mul ( M_, temp, M_ );

	}

	printf("\n[%4.2f] M and M_ done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	clearLongVariable( $SQR_M_MOD_N );
	clearLongVariable( $InverseByModuleMM_ );
	clearLongVariable( $InverseByModule$NM );
	clearLongVariable( temp );
	
	// $SQR_M_MOD_N = M*M % $N;
	//mul ( M, M, temp );
	//mod (temp,  $N,  $SQR_M_MOD_N); 
	 
	
	//sqrMmodN(temp, $N, $SQR_M_MOD_N);

	mod (M,  $N,  temp);
	mul (temp,  temp,  $SQR_M_MOD_N); 
	copyVariable($SQR_M_MOD_N,temp);
	mod (temp,$N,$SQR_M_MOD_N);

	

	printf("\n[%4.2f] $SQR_M_MOD_N done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	InverseByModule(M, M_, $InverseByModuleMM_);
	printf("\n[%4.2f] $InverseByModuleMM_ done [%i iterations].\n",(double)(clock() - begin) / CLOCKS_PER_SEC, profiler_extended_euclid_Iterations[0]);
	
	InverseByModule ($N, M , $InverseByModule$NM);
	printf("\n[%4.2f] $InverseByModule$NM done [%i iterations].\n",(double)(clock() - begin) / CLOCKS_PER_SEC, profiler_extended_euclid_Iterations[1]);

	for( int i = 0; i < THREAD_NUMBER; i++){
			 
		ABmodN[0][i] = 1;
		ABmodN[1][i] = 1;
		
		mod_($N, Base[i], &N[i]);
		mod_($N, Base_[i], &N_[i]);
		
		div_(M, Base[i], Mi[i]);
		div_(M_, Base_[i], Mi_[i]);
		
		InverseByModule_(Mi[i], Base[i], &MiInv[i]);
		InverseByModule_(Mi_[i], Base_[i], &Mi_Inv[i]);
		
		mod_($SQR_M_MOD_N, Base[i], &SQR_M_MOD_N_RNS[i]);
		mod_($SQR_M_MOD_N, Base_[i], &SQR_M_MOD_N_RNS_[i]);

		mod_($InverseByModuleMM_, Base_[i], &InverseByModuleMM_[i]);
		mod_($InverseByModule$NM, Base[i], &InverseByModuleNM[i]);

		mod_(Mi_[i], Me, &Mi_InMe[i]);
		mod_(Mi[i], Me, &MiInMe[i]);

		mod_(M_, Base[i], &M_InBase[i]);

		printf("\n[%4.2f] #%i thread computation done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC, i);
	}


	for (int i = 0; i < THREAD_NUMBER; i++){
		for (int j = 0; j < THREAD_NUMBER; j++){

			mod_( Mi [j], Base_ [i], &MiInBase_[i][j] );
			mod_( Mi_ [j], Base [i], &Mi_InBase[i][j] );

		}
	}

	printf("\n[%4.2f] &MiInBase_ and &Mi_InBase done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	mod_($N, Me, &N_Me);

	clearLongVariable( temp );
	
	//(M_^-1 mod Me) % Me;
	unsigned int t;
	InverseByModule_(M_, Me, &t);
	temp[ MAX - 1] = t;
	mod_(temp, Me, &InverseByModuleM_Me);
	
	printf("\n[%4.2f] &InverseByModuleM_Me done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC);


	//(( M_ * Me ) ^ -1 mod Me) % Me;
	unsigned int tempM_Me[ MAX ];
	clearLongVariable( tempM_Me );
	clearLongVariable( temp );
	temp[MAX - 1] = Me;
	mul(M_, temp, tempM_Me);
	clearLongVariable( temp );
	InverseByModule(M, tempM_Me, temp);
	mod_(temp, Me, &InverseByModuleMMMe);
	
	printf("\n[%4.2f] &InverseByModuleMMMe done.\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	printf("\nTOTAL TIME SPENT: %f s\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

}


int main(int argc, char **argv)
{
	
	//argv[1] - input message
	//argv[2] - action message
	//argv[3] - method message

	int mode; //encr/decr
	int unit; //CPU/GPU

	unsigned int $d[MAX];
	unsigned int $e[MAX];
	unsigned int q[MAX];
	unsigned int p[MAX];
	//unsigned int n[MAX];
	unsigned int p_minus_1[MAX];
	unsigned int q_minus_1[MAX];
	unsigned int phi[MAX];
	unsigned int temp[ MAX ];
	unsigned int $input[ MAX ];
	unsigned int R[THREAD_NUMBER], R_[THREAD_NUMBER];
	clock_t begin;
	hipError_t cudaStatus;
	char output [ MAX * 10 ];

	if(
	   argv[2][0] == *("e") && 
	   argv[2][1] == *("n") && 
	   argv[2][2] == *("c") && 
	   argv[2][3] == *("r") && 
	   argv[2][4] == *("y") && 
	   argv[2][5] == *("p") && 
	   argv[2][6] == *("t") ) mode = 1;
   else if( 
	   argv[2][0] == *("d") && 
	   argv[2][1] == *("e") && 
	   argv[2][2] == *("c") && 
	   argv[2][3] == *("r") && 
	   argv[2][4] == *("y") && 
	   argv[2][5] == *("p") && 
	   argv[2][6] == *("t") ) mode = 2;
   else {
	   printf("Wrong mode: must be \"encrypt\" or \"decrypt\" (case sensitive) ");
	   goto Error;
   }
	
   if( argv[3][0] == *("C") && 
	   argv[3][1] == *("P") && 
	   argv[3][2] == *("U")) unit = 1;
   else if( 
	   argv[3][0] == *("G") && 
	   argv[3][1] == *("P") && 
	   argv[3][2] == *("U")) unit = 2;
   else {
	   printf("Wrong unit: must be \"CPU\" or \"GPU\" (case sensitive) ");
	   goto Error;
   }

  


	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

	
	/*// n = p*q
	mul ( p, q, n );

	clearLongVariable (  _1 );
	_1[ MAX - 1 ] = 1;		// for storing digit "1"
	clearLongVariable ( p_minus_1 );
	clearLongVariable ( q_minus_1 );

	// phi = (p-1)*(q-1);
	sub ( p, _1, p_minus_1 );
	sub ( q, _1, q_minus_1 );
	mul ( p_minus_1, q_minus_1, phi );

	

	*/
	
	Me = 64;
	/*
	convertToNormalForm( "130ebebd67b16a9ab2c53a437badbf8f01a80c750095a7fcfe95742c3d5ed1abb318babc5cb5d9350fee4da65ee074f65e1758117e6945f0fcfc8137528053ce9d1da8618890dee24e5e0bf8c87795bb1d09eddd544640824ee0dd0ea9fd908d27b0f8a1ae5c37f3647fbf2f5795500ad76c195b3387d0458a8f51b701472301" , $N);      // modulus
	convertToNormalForm( "0123" , $input);  // message
	convertToNormalForm( "010001" , $e); // public exponenta
	convertToNormalForm( "12e8da920d4599458e84ec5ef1656161807f427d05eb79182b7418259d6f6c14364d1f5caf9130c8d9d9d6ea71d1bdbc87781a46a16bcb9e672814fed3b9c96ddffe0a1b0955ae68055c8f92fef518a04fc32a2ea8390e617cc5556a251f9ae9eee70a32e579cb3e9f298848a9b3aaf634f5930ffbf74473f7cb6c0cefee1751" , $d); // secret exponenta 
	*/
	
	
	convertToNormalForm( "025123" , $N);      // modulus
	
	convertToNormalForm( "01365D" , $e); // public exponenta
	convertToNormalForm( "0AD" , $d) ; // secret exponenta 
	

	convertPowerToArrayOfBits($e, e, &eBitsCount, &numberOfIterationsForE);
	convertPowerToArrayOfBits($d, d, &dBitsCount, &numberOfIterationsForD);

	for (int i = 0; i < MAX; i++) {
		ZERO_MEMORY[ i ] = 0; 
	}

	

	for(int i=0; i<THREAD_NUMBER;i++){
		Base[i] = 4294967296 - HHH[i];
		Base_[i] = 4294967296 - KKK[i];
	}


	

	generateStaticData();
	

	convertToNormalForm( argv[1] , $input);  // message

	for(int z=0; z< THREAD_NUMBER;z++) {
		mod_($input, Base[z], &input[z]);
		mod_($input, Base_[z], &input_[z]);
	}


	if( mode == 1 ){

	//encrypt
	iterationsCounter = 1;
	currentOperationIndex = 0;
	begin = clock();
	for (int z = MAX * INT_SIZE - eBitsCount; z < MAX * INT_SIZE; z++){

		MM(ABmodN[0], ABmodN[1], ABmodN[0], ABmodN[1], R, R_);
		MM(R, R_, SQR_M_MOD_N_RNS, SQR_M_MOD_N_RNS_, ABmodN[0], ABmodN[1]);

		if (e[z]){

			MM(ABmodN[0], ABmodN[1], input, input_, R, R_);
			MM(R, R_, SQR_M_MOD_N_RNS, SQR_M_MOD_N_RNS_, ABmodN[0], ABmodN[1]);

		} 
	}
	if(unit == 2)
		printf("\nEncryption ended in : %f s\n",(double)(clock() - begin) / CLOCKS_PER_SEC / 15);
	else
		printf("\nEncryption ended in : %f s\n",(double)(clock() - begin) / CLOCKS_PER_SEC);

	};
	
	if( mode == 2 ){

	
	//decrypt
	iterationsCounter = 1;
	currentOperationIndex = 1;
	begin = clock();
	for (int z = MAX * INT_SIZE - dBitsCount; z < MAX * INT_SIZE; z++){

		MM(ABmodN[0], ABmodN[1], ABmodN[0], ABmodN[1], R, R_);
		MM(R, R_, SQR_M_MOD_N_RNS, SQR_M_MOD_N_RNS_, ABmodN[0], ABmodN[1]);

		if (d[z]){

			MM(ABmodN[0], ABmodN[1], input, input_, R, R_);
			MM(R, R_, SQR_M_MOD_N_RNS, SQR_M_MOD_N_RNS_, ABmodN[0], ABmodN[1]);

		} 
	}

	if(unit == 2)
		printf("\nDecryption ended in : %f s\n",(double)(clock() - begin) / CLOCKS_PER_SEC / 15);
	else
		printf("\nDecryption ended in : %f s\n",(double)(clock() - begin) / CLOCKS_PER_SEC);
	


	}

	convertFromRNS(ABmodN[0], temp);
	convertFromNormalForm( output, temp );
	printf("%s\n", output);


	showProfilerInfo();

	/*

	const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };
    
	
    // Add vectors in parallel.
   
	for (int i=0; i< 100000; i++) {
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
	}

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   /* cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	*/
Error:
	getch();
    return 0;
}








